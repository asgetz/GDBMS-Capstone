
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCKSIZE 256

/**********/
/* iDivUp */
/**********/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/***********************/
/* CUDA ERROR CHECKING */
/***********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/********************************************************/
/* KERNEL FUNCTION FOR TESTING RANDOM NUMBER GENERATION */
/********************************************************/
__global__ void testrand1(unsigned long seed, float *a, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    if (idx < N) {
        hiprand_init(seed, idx, 0, &state);
        a[(idx*2)] = hiprand_uniform(&state);
        if(idx%2)
          skipahead_sequence(1, &state);
        a[(idx*2)+1] = hiprand_uniform(&state);

    }
}

/********/
/* MAIN */
/********/
int main() {

    const int N = 10;

    float *h_a  = (float*)malloc(2*N*sizeof(float));
    float *d_a; gpuErrchk(hipMalloc((void**)&d_a, 2*N*sizeof(float)));

    testrand1<<<iDivUp(N, BLOCKSIZE), BLOCKSIZE>>>(1235, d_a, N);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_a, d_a, 2*N*sizeof(float), hipMemcpyDeviceToHost));

    for (int i=0; i<2*N; i++) printf("%i %f\n", i, h_a[i]);

}
