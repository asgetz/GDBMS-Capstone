/*
 * This program uses the device CURAND API to calculate what
 * proportion of pseudo-random ints have low bit set.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void generate_kernel(hiprandStateMtgp32 *state,
                                int n,
                                int *result)
{
    int id = threadIdx.x + blockIdx.x * 256;
    int count = 0;
    unsigned int x;
    /* Generate pseudo-random unsigned ints */
    for(int i = 0; i < n; i++) {
        x = hiprand(&state[blockIdx.x]);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Store results */
    result[id] += count;
}

int main(int argc, char *argv[])
{
    int i;
    long long total;
    hiprandStateMtgp32 *devMTGPStates;
    mtgp32_kernel_params *devKernelParams;
    int *devResults, *hostResults;
    int sampleCount = 10000;

    /* Allow over-ride of sample count */
    if (argc == 2) {
        sscanf(argv[1],"%d",&sampleCount);
    }

    /* Allocate space for results on host */
    hostResults = (int *)calloc(64 * 256, sizeof(int));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 256 *
              sizeof(int)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 256 *
              sizeof(int)));

    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devMTGPStates, 64 *
              sizeof(hiprandStateMtgp32)));

    /* Setup MTGP prng states */

    /* Allocate space for MTGP kernel parameters */
    CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));

    /* Reformat from predefined parameter sets to kernel format, */
    /* and copy kernel parameters to device memory               */
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));

    /* Initialize one state per thread block */
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates,
                mtgp32dc_params_fast_11213, devKernelParams, 64, 1234));

    /* State setup is complete */

    /* Generate and use pseudo-random  */
    for(i = 0; i < 10; i++) {
        generate_kernel<<<64, 256>>>(devMTGPStates, sampleCount, devResults);
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 256 *
        sizeof(int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 256; i++) {
        total += hostResults[i];
    }


    printf("Fraction with low bit set was %10.13g\n",
        (double)total / (64.0f * 256.0f * sampleCount * 10.0f));

    /* Cleanup */
    CUDA_CALL(hipFree(devMTGPStates));
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_mtgp_example PASSED\n");
    return EXIT_SUCCESS;
}
