/////////////////////////


#include <hip/hip_runtime.h>
#include <stdio.h>  /* Enables printing output to console */
#define N 64	/* Speficy array length value */
#define TPB 32  /* Threads per block used in kernel */

__device__
float scale(int i, int n){ return ((float)i)/(n-1); }

__device__
float distance(float x1, float x2){
	return sqrt((x2-x1)*(x2-x1));
}

__global__
void distanceKernel(float *d_out, float ref, int len){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x=scale(i,len);
	d_out[i]=distance(x,ref);
	printf("i = %2d: dist from %f to %f is %f.\n",i,ref,x,d_out[i]);
}

int main(){
	const float ref=0.5f;

	//Declare a pointer for an array of floats
	float *d_out=0;

	//Allocate device memory to store the output array
	hipMalloc(&d_out, N*sizeof(float));

	//Launch kernel to compute and store distance values
	distanceKernel<<<N/TPB,TPB>>>(d_out,ref,N);

	hipFree(d_out);

	return 0;
}
