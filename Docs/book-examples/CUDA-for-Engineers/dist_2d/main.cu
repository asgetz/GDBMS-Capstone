
#include <hip/hip_runtime.h>
#define W 500
#define H 500
#define TX 32 // number of threads per block along x-axis
#define TY 32 // number of threads per block along y-axis

__global__ void distanceKernel(float *d_out, int w, int h, float2 pos) 
{
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;
  const int i=r*w+c;

  if ((c >= w) || (r >= h)) return; 

  // Compute the distance and set d_out[i]
  d_out[i] = sqrtf((c - pos.x)*(c - pos.x) + (r - pos.y)*(r - pos.y));
}

int main() 
{
  float *out = (float*)calloc(W*H, sizeof(float));
  float *d_out; // pointer for device array

  hipMalloc(&d_out,W*H*sizeof(float));

  const float2 pos = {0.0f, 0.0f}; // set reference position
  const dim3 blockSize(TX, TY);
  const int bx=(W+TX-1)/TX;
  const int by=(W+TY-1)/TY;
  const dim3 gridSize = dim3(bx,by);

  distanceKernel<<<gridSize, blockSize>>>(d_out, W, H, pos);

  // Copy results to host.
  hipMemcpy(out, d_out, W*H*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_out);
  free(out);
  return 0;
}