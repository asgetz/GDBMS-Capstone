#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32
#define RAD 1

int divUp(int a, int b) { return (a + b - 1)/b; }

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__device__
int idxClip(int idx, int idxMax) {
  return idx >(idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx);
}

__device__
int flatten(int col, int row, int width, int height) {
  return idxClip(col, width) + idxClip(row, height)*width;
}

__global__
void sharpenKernel(uchar4 *d_out, const uchar4 *d_in,
                   const float *d_filter, int w, int h) {
  const int c = threadIdx.x + blockDim.x * blockIdx.x;
  const int r = threadIdx.y + blockDim.y * blockIdx.y;
  if ((c >= w) || (r >= h)) return;
  const int i = flatten(c, r, w, h);
  const int s_c = threadIdx.x + RAD;
  const int s_r = threadIdx.y + RAD;
  const int s_w = blockDim.x + 2 * RAD;
  const int s_h = blockDim.y + 2 * RAD;
  const int s_i = flatten(s_c, s_r, s_w, s_h);
  const int fltSz = 2 * RAD + 1;
  
  extern __shared__ uchar4 s_block[];
  uchar4 *s_in = s_block;
  uchar4 *s_out = &s_block[s_w*s_h];
  
  // Regular cells
  s_in[s_i] = d_in[i];
  
  // Halo cells
  if (threadIdx.x < RAD && threadIdx.y < RAD) {
    s_in[flatten(s_c - RAD, s_r - RAD, s_w, s_h)] =
      d_in[flatten(c - RAD, r - RAD, w, h)];
    s_in[flatten(s_c + blockDim.x, s_r - RAD, s_w, s_h)] =
      d_in[flatten(c + blockDim.x, r - RAD, w, h)];
    s_in[flatten(s_c - RAD, s_r + blockDim.y, s_w, s_h)] =
      d_in[flatten(c - RAD, r + blockDim.y, w, h)];
    s_in[flatten(s_c + blockDim.x, s_r + blockDim.y, s_w, s_h)] =
      d_in[flatten(c + blockDim.x, r + blockDim.y, w, h)];
  }
  if (threadIdx.x < RAD) {
    s_in[flatten(s_c - RAD, s_r, s_w, s_h)] =
      d_in[flatten(c - RAD, r, w, h)];
    s_in[flatten(s_c + blockDim.x, s_r, s_w, s_h)] =
      d_in[flatten(c + blockDim.x, r, w, h)];
  }
  if (threadIdx.y < RAD) {
    s_in[flatten(s_c, s_r - RAD, s_w, s_h)] =
      d_in[flatten(c, r - RAD, w, h)];
    s_in[flatten(s_c, s_r + blockDim.y, s_w, s_h)] =
      d_in[flatten(c, r + blockDim.y, w, h)];
  }
  __syncthreads();
  
  float rgb[3] = { 0.f, 0.f, 0.f };
  for (int rd = -RAD; rd <= RAD; ++rd) {
    for (int cd = -RAD; cd <= RAD; ++cd) {
      const int s_imgIdx = flatten(s_c + cd, s_r + rd, s_w, s_h);
      const int fltIdx = flatten(RAD + cd, RAD + rd, fltSz, fltSz);
      const uchar4 color = s_in[s_imgIdx];
      const float weight = d_filter[fltIdx];
      rgb[0] += weight*color.x;
      rgb[1] += weight*color.y;
      rgb[2] += weight*color.z;
    }
  }

  const int s_outIdx = threadIdx.y*blockDim.x + threadIdx.x;
  s_out[s_outIdx].x = clip(rgb[0]);
  s_out[s_outIdx].y = clip(rgb[1]);
  s_out[s_outIdx].z = clip(rgb[2]);
  
  __syncthreads();
  d_out[i] = s_out[s_outIdx];
}

void sharpenParallel(uchar4 *arr, int w, int h) {
  const int fltSz = 2*RAD + 1;
  const float filter[9] = {-1.0, -1.0, -1.0,
                           -1.0,  9.0, -1.0,
                           -1.0, -1.0, -1.0};
  uchar4 *d_in = 0, *d_out = 0;
  float *d_filter = 0;
  
  hipMalloc(&d_in, w*h*sizeof(uchar4));
  hipMemcpy(d_in, arr, w*h*sizeof(uchar4), hipMemcpyHostToDevice);
  hipMalloc(&d_out, w*h*sizeof(uchar4));
  hipMalloc(&d_filter, fltSz*fltSz*sizeof(float));
  hipMemcpy(d_filter, filter, fltSz*fltSz*sizeof(float),
             hipMemcpyHostToDevice);

  const dim3 blockSize(TX, TY);
  const dim3 gridSize(divUp(w, TX), divUp(h, TY));
  const size_t smSz = ((TX+2*RAD)*(TY+2*RAD)+(TX*TY))*sizeof(uchar4);
  sharpenKernel<<<gridSize, blockSize, smSz>>>(d_out, d_in, d_filter, 
                                               w, h);
  hipMemcpy(arr, d_out, w*h*sizeof(uchar4), hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_filter);
}