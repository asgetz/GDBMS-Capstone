#include "hip/hip_runtime.h"
#include "kernel.h"
#include "device_funcs.cuh"
#include <hip/hip_vector_types.h>
#define TX_2D 32
#define TY_2D 32
#define TX 8
#define TY 8
#define TZ 8
#define NUMSTEPS 20

__global__
void renderKernel(uchar4 *d_out, float *d_vol, int w, int h,
  int3 volSize, int method, float zs, float theta, float threshold,
  float dist) {
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;
  const int i = c + r * w;
  if ((c >= w) || (r >= h)) return; // Check if within image bounds
  const uchar4 background = { 64, 0, 128, 0 };
  float3 source = { 0.f, 0.f, -zs };
  float3 pix = scrIdxToPos(c, r, w, h, 2 * volSize.z - zs);
  // apply viewing transformation: here rotate about y-axis
  source = yRotate(source, theta);
  pix = yRotate(pix, theta);
  // prepare inputs for ray-box intersection
  float t0, t1;
  const Ray pixRay = {source, pix - source};
  float3 center = {volSize.x/2.f, volSize.y/2.f, volSize.z/2.f};
  const float3 boxmin = -center;
  const float3 boxmax = {volSize.x - center.x, volSize.y - center.y,
                         volSize.z - center.z};
  // perform ray-box intersection test
  const bool hitBox = intersectBox(pixRay, boxmin, boxmax, &t0, &t1);
  uchar4 shade;
  if (!hitBox) shade = background; //miss box => background color
  else {
    if (t0 < 0.0f) t0 = 0.f; // clamp to 0 to avoid looking backward
    // bounded by points where the ray enters and leaves the box
    const Ray boxRay = { paramRay(pixRay, t0),
    paramRay(pixRay, t1) - paramRay(pixRay, t0) };
    if (method == 1) shade = 
      sliceShader(d_vol, volSize, boxRay, threshold, dist, source);
    else if (method == 2) shade =
      rayCastShader(d_vol, volSize, boxRay, threshold);
    else shade =
      volumeRenderShader(d_vol, volSize, boxRay, threshold, NUMSTEPS);
  }
  d_out[i] = shade;
}

__global__
void volumeKernel(float *d_vol, int3 volSize, int id, float4 params) {
  const int w = volSize.x, h = volSize.y, d = volSize.z;
  const int c = blockIdx.x * blockDim.x + threadIdx.x; // column
  const int r = blockIdx.y * blockDim.y + threadIdx.y; // row
  const int s = blockIdx.z * blockDim.z + threadIdx.z; // stack
  const int i = c + r * w + s * w * h;
  if ((c >= w) || (r >= h) || (s >= d)) return;
  d_vol[i] = func(c, r, s, id, volSize, params); // compute and store
}

void kernelLauncher(uchar4 *d_out, float *d_vol, int w, int h,
  int3 volSize, int method, int zs, float theta, float threshold,
  float dist) {
  dim3 blockSize(TX_2D, TY_2D);
  dim3 gridSize(divUp(w, TX_2D), divUp(h, TY_2D));
  renderKernel<<<gridSize, blockSize>>>(d_out, d_vol, w, h, volSize,
    method, zs, theta, threshold, dist);
}

void volumeKernelLauncher(float *d_vol, int3 volSize, int id,
  float4 params) {
  dim3 blockSize(TX, TY, TZ);
  dim3 gridSize(divUp(volSize.x, TX), divUp(volSize.y, TY),
  divUp(volSize.z, TZ));
  volumeKernel<<<gridSize, blockSize>>>(d_vol, volSize, id, params); 
}