#include "hip/hip_runtime.h"
#include "device_funcs.cuh"
#include <hip/hip_vector_types.h>
#define EPS 0.01f

__host__ int divUp(int a, int b) { return (a + b - 1)/b; }

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__device__ int clipWithBounds(int n, int n_min, int n_max) {
  return n > n_max ? n_max : (n < n_min ? n_min : n);
}

__device__ float3 yRotate(float3 pos, float theta) {
  const float c = cosf(theta), s = sinf(theta);
  return make_float3(c*pos.x + s*pos.z, pos.y, -s*pos.x + c*pos.z);
}

__device__ float func(int c, int r, int s, int id, int3 volSize,
  float4 params) {
  const int3 pos0 = { volSize.x / 2, volSize.y / 2, volSize.z / 2 };
  const float dx = c - pos0.x, dy = r - pos0.y, dz = s - pos0.z;
  // sphere
  if (id == 0) { return sqrtf(dx*dx + dy*dy + dz*dz) - params.x; }
  else if (id == 1) { // torus
    const float r = sqrtf(dx*dx + dy*dy);
    return sqrtf((r - params.x)*(r - params.x) + dz*dz) - params.y;
  }
  else { // block
    float x = fabsf(dx) - params.x, y = fabsf(dy) - params.y,
          z = fabsf(dz) - params.z;
    if (x <= 0 && y <= 0 && z <= 0) return fmaxf(x, fmaxf(y, z));
    else {
      x = fmaxf(x, 0), y = fmaxf(y, 0), z = fmaxf(z, 0);
      return sqrtf(x*x + y*y + z*z);
    }
  }
}

__device__ float3 scrIdxToPos(int c, int r, int w, int h, float zs) {
  return make_float3(c - w / 2, r - h / 2, zs);
}

__device__ float3 paramRay(Ray r, float t) { return r.o + t*(r.d); }

__device__ float planeSDF(float3 pos, float3 norm, float d) {
  return dot(pos, normalize(norm)) - d;
}

__device__
bool rayPlaneIntersect(Ray myRay, float3 n, float dist, float *t) {
  const float f0 = planeSDF(paramRay(myRay, 0.f), n, dist);
  const float f1 = planeSDF(paramRay(myRay, 1.f), n, dist);
  bool result = (f0*f1 < 0);
  if (result) *t = (0.f - f0) / (f1 - f0);
  return result;
}

// Intersect ray with a box from volumeRender SDK sample.
__device__ bool intersectBox(Ray r, float3 boxmin, float3 boxmax,
  float *tnear, float *tfar) {
  // Compute intersection of ray with all six bbox planes.
  const float3 invR = make_float3(1.0f) / r.d;
  const float3 tbot = invR*(boxmin - r.o), ttop = invR*(boxmax - r.o);
  // Re-order intersections to find smallest and largest on each axis.
  const float3 tmin = fminf(ttop, tbot), tmax = fmaxf(ttop, tbot);
  // Find the largest tmin and the smallest tmax.
  *tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
  *tfar = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));
  return *tfar > *tnear;
}

__device__ int3 posToVolIndex(float3 pos, int3 volSize) {
  return make_int3(pos.x + volSize.x/2, pos.y + volSize.y/2,
                   pos.z + volSize.z/2);
}

__device__ int flatten(int3 index, int3 volSize) {
  return index.x + index.y*volSize.x + index.z*volSize.x*volSize.y;
}

__device__ float density(float *d_vol, int3 volSize, float3 pos) {
  int3 index = posToVolIndex(pos, volSize);
  int i = index.x, j = index.y, k = index.z;
  const int w = volSize.x, h = volSize.y, d = volSize.z;
  const float3 rem = fracf(pos);
  index = make_int3(clipWithBounds(i, 0, w - 2),
    clipWithBounds(j, 0, h - 2), clipWithBounds(k, 0, d - 2));
  // directed increments for computing the gradient
  const int3 dx = { 1, 0, 0 }, dy = { 0, 1, 0 }, dz = { 0, 0, 1 };
  // values sampled at surrounding grid points
  const float dens000 = d_vol[flatten(index, volSize)];
  const float dens100 = d_vol[flatten(index + dx, volSize)];
  const float dens010 = d_vol[flatten(index + dy, volSize)];
  const float dens001 = d_vol[flatten(index + dz, volSize)];
  const float dens110 = d_vol[flatten(index + dx + dy, volSize)];
  const float dens101 = d_vol[flatten(index + dx + dz, volSize)];
  const float dens011 = d_vol[flatten(index + dy + dz, volSize)];
  const float dens111 = d_vol[flatten(index + dx + dy + dz, volSize)];
  // trilinear interpolation
  return (1 - rem.x)*(1 - rem.y)*(1 - rem.z)*dens000 +
    (rem.x)*(1 - rem.y)*(1 - rem.z)*dens100 +
    (1 - rem.x)*(rem.y)*(1 - rem.z)*dens010 +
    (1 - rem.x)*(1 - rem.y)*(rem.z)*dens001 +
    (rem.x)*(rem.y)*(1 - rem.z)*dens110 +
    (rem.x)*(1 - rem.y)*(rem.z)*dens101 +
    (1 - rem.x)*(rem.y)*(rem.z)*dens011 +
    (rem.x)*(rem.y)*(rem.z)*dens111;
}

__device__ uchar4 sliceShader(float *d_vol, int3 volSize, Ray boxRay,
  float gain, float dist, float3 norm) {
  float t;
  uchar4 shade = make_uchar4(96, 0, 192, 0); // background value
  if (rayPlaneIntersect(boxRay, norm, dist, &t)) {
    float sliceDens = density(d_vol, volSize, paramRay(boxRay, t));
    shade = make_uchar4(48, clip(-10.f * (1.0f + gain) * sliceDens),
                        96, 255);
  }
  return shade;
}

__device__ uchar4 volumeRenderShader(float *d_vol, int3 volSize,
  Ray boxRay, float threshold, int numSteps) {
  uchar4 shade = make_uchar4(96, 0, 192, 0); // background value
  const float dt = 1.f / numSteps;
  const float len = length(boxRay.d) / numSteps;
  float accum = 0.f;
  float3 pos = boxRay.o;
  float val = density(d_vol, volSize, pos);
  for (float t = 0.f; t<1.f; t += dt) {
    if (val - threshold < 0.f) accum += (fabsf(val - threshold))*len;
    pos = paramRay(boxRay, t);
    val = density(d_vol, volSize, pos); 
  }
  if (clip(accum) > 0.f) shade.y = clip(accum);
  return shade;
}

__device__ uchar4 rayCastShader(float *d_vol, int3 volSize,
  Ray boxRay, float dist) {
  uchar4 shade = make_uchar4(96, 0, 192, 0);
  float3 pos = boxRay.o;
  float len = length(boxRay.d);
  float t = 0.0f;
  float f = density(d_vol, volSize, pos);
  while (f > dist + EPS && t < 1.0f) {
    f = density(d_vol, volSize, pos);
    t += (f - dist) / len;
    pos = paramRay(boxRay, t);
    f = density(d_vol, volSize, pos);
  }
  if (t < 1.f) {
    const float3 ux = make_float3(1, 0, 0), uy = make_float3(0, 1, 0),
                 uz = make_float3(0, 0, 1);
    float3 grad = {(density(d_vol, volSize, pos + EPS*ux) -
                    density(d_vol, volSize, pos))/EPS,
                   (density(d_vol, volSize, pos + EPS*uy) -
                   density(d_vol, volSize, pos))/EPS,
                   (density(d_vol, volSize, pos + EPS*uz) -
                   density(d_vol, volSize, pos))/EPS};
    float intensity = -dot(normalize(boxRay.d), normalize(grad));
    shade = make_uchar4(255 * intensity, 0, 0, 255);
  }
  return shade;
}
