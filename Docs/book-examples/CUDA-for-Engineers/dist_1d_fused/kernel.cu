#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>
#include <math.h>
#include <stdio.h>
#define N 64

// DistanceFrom(ref,n)(x)->sqrt((x/(n-1)-ref)*(x/(n-1)-ref))
struct DistanceFrom {
  DistanceFrom(float ref, int n) : mRef(ref), mN(n) {}
   
  __host__ __device__
  float operator()(const float &x) {
    float scaledX = x / (mN - 1);
    return std::sqrt((scaledX - mRef)*(scaledX - mRef));
  }
  float mRef;
  int mN;
};

int main() {
  const float ref = 0.5;
  thrust::device_vector<float> dvec_dist(N);
  thrust::transform(thrust::counting_iterator<float>(0),
    thrust::counting_iterator<float>(N), dvec_dist.begin(),
    DistanceFrom(ref, N));
  
  thrust::host_vector<float> hvec_dist = dvec_dist;
  float *ptr = thrust::raw_pointer_cast(&hvec_dist[0]); // debugging
  for (int i = 0; i < N; ++i) {
    printf("x[%d]=%.3f, dist=%.3f\n", i, 1.f*i / (N - 1), hvec_dist[i]);
  }
  return 0;
}