// kernel.cu
// Alex Getz

#include <cstddef>
#include <stdexcept>
#include <memory>
#include <hip/hip_runtime_api.h>

#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <algorithm>
#include <iostream>
#include <sstream>
#include <cstring>
#include <string>

#include <random>
#include <functional>
#include <math.h>
#include <algorithm>
#include <array>

#define MAX_ENTRIES 11897026
#define B_SIZE 2000
#define TPB 128
#define SHMEM_SIZE 1024 * sizeof(int)








///////////////////////////////////////////////////////////////////////////////////////
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if(result!=cudaSuccess){
        fprintf(stderr,"CUDA Runtime Error: %s\n",
                cudaGetErrorString(result));
        assert(result==cudaSuccess);
    }
#endif
    return result;
}

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} /*while(0)*/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__);}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess){
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}
////////////////////////////////////////////////////////////////////////////////////////





/* ///   DEVICE FUNCTIONS   /// */

__device__ float getnextrand(hiprandState *state){
    return (float)(hiprand_uniform(state));
}

__device__ int getnextrandscaled(hiprandState *state, unsigned long int scale){
    return (unsigned long int) scale * getnextrand(state);
}


/* ///   DEVICE KERNELS   /// */

__global__ void initCurand(hiprandState *state, unsigned long seed){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(idx+seed, 0, 0, &state[idx]);
}

__global__ void bootstrap(double *output_mean, int *d_sample, hiprandState *state)
{
    __shared__ int partial_Sums[SHMEM_SIZE];

    unsigned int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tNum = threadIdx.x;
    int bNum = blockIdx.x;
    int bSize = blockDim.x;
    unsigned long int ts = 0;
    long long int tSum = 0;
    int count = 0;

    for(unsigned int i=tNum; i<MAX_ENTRIES; i+=bSize){
	ts = getnextrandscaled(&state[idx], MAX_ENTRIES);
	tSum += d_sample[ts];
	count++;
    }
    partial_Sums[tNum] = tSum / count;
    __syncthreads();

    // Perform Sum reduction across all the threads of the block
    for(int s=(bSize/2); s>0; s >>= 1){
	// Each thread does work unless the index goes off the block
	if(tNum<s){
	    partial_Sums[tNum] += partial_Sums[tNum+s];
	}
	__syncthreads();
    }

    // Use first thread of each block to write results back to main mem
    if(tNum==0){
        output_mean[bNum] = (double)partial_Sums[0] / (double)bSize;
    }
}



/* ///   HOST GLOBAL FUNCTIONS   /// */

void throw_error(hipError_t err){
    if(err != hipSuccess)
	throw std::runtime_error(hipGetErrorString(err));
}


/* ///   HOST GLOBAL STRUCTS & VARIABLES   /// */

struct cuda_free_deleter_t{
    void operator()(void* ptr) const
    {
	hipFree(ptr);
    }
};


template <typename T>
auto cudaAllocBuffer(std::size_t size){
    void *ptr;
    throw_error(hipMalloc(&ptr, size*sizeof(T)));
    return std::unique_ptr<T, cuda_free_deleter_t> { static_cast<T*>(ptr) };
}


int main(){
    int *BaseSample, *d_Base;
    double *d_mean, *h_mean;
    //curandState *devStates;
    checkCuda( hipHostMalloc((void**)&BaseSample,MAX_ENTRIES*sizeof(int), hipHostMallocDefault));
    //checkCuda( cudaMalloc((void**)&devStates,2048*1024*sizeof(curandState)));

    std::string line;
    uintmax_t m_numLines = 0;
    std::ifstream fs("../data/allCountries.txt");
    if(!fs){
        std::cout<<"ERROR\n";
    }else{
        while (std::getline(fs, line))
        {
            int counter=0;
            std::stringstream ss;
            std::string temp;
            // std::cout<<"\n"<<line<<"\n";
            ss << line;
            std::getline(ss,temp,'\t');
            // std::cout<<temp<<", position: "<<++counter<<"\n";
            while(std::getline(ss,temp,'\t')){
                if(temp.length() == 4){
                    BaseSample[m_numLines] = std::atoi(temp.c_str());



                    // std::cout<<temp<<", position: "<<++counter<<"\n";
                    break;
                } else{ ++counter; }
            }
            m_numLines++;
            // if(m_numLines==5){ break; }
        }
    }
    //std::cout << "m_numLines = " << m_numLines << "\nMoving on...\n\n";
    fs.close();

    //std::cout << "Element 300,000 of BaseSample: " << BaseSample[300000]<<std::endl;


    ///---			Calculating the Mean			 ---///
    ///////////////////////////////////////////////////////////////////////////
    std::cout<<"Sample has been generated.\nCalculating the mean...\n";
    long long int BaseSum = 0;
    for(int i=0;i<MAX_ENTRIES;i++){
	BaseSum += BaseSample[i];
    }
    double BaseMean = (double)BaseSum / (double)MAX_ENTRIES;
    std::cout<<"Mean has been calculated! Moving on...\n\n";
    


    ///---			KERNEL OPERATIONS			 ---///
    ///////////////////////////////////////////////////////////////////////////
    checkCuda( hipMalloc((void**)&d_Base,MAX_ENTRIES*sizeof(int)));
    checkCuda( hipMemcpy(d_Base,BaseSample,MAX_ENTRIES*sizeof(int),hipMemcpyHostToDevice));


    //checkCuda( cudaFreeHost(BaseSample) );

    checkCuda( hipMalloc((void**)&d_mean,2048*sizeof(double)));
    checkCuda( hipHostMalloc((void**)&h_mean,2048*sizeof(double), hipHostMallocDefault));

    std::cout<<"Launching initCurand Kernel now\n\n";

    //////////////////////////////////////
    //checkCuda( cudaProfilerStart() );
    try{
	constexpr int block_size = 512;
	constexpr int num_blocks = 4096;
	auto devStates = cudaAllocBuffer<hiprandState>(num_blocks * block_size);
	initCurand<<<num_blocks, block_size>>>(devStates.get(),1234);
	throw_error(hipPeekAtLastError());
	throw_error(hipDeviceSynchronize());
	std::cout<<"Curand Kernel Launch Try block SUCCESSFUL!\n";
	std::cout<<"Launching Bootstrap Kernel now\n\n";
	bootstrap<<<2048,1024>>>(d_mean,d_Base,devStates.get());
	throw_error(hipPeekAtLastError());
	throw_error(hipDeviceSynchronize());
	std::cout<<"Bootstrap Kernel Launch Try Block SUCCESSFUL!\n";
    }
    catch (const std::exception& e)
    {
	std::cerr << "Error: " << e.what() << '\n';
	return -1;
    }
    catch (...)
    {
	std::cerr << "Unknown Exception";
	return -1;
    }

    std::cout<<"Kernels appear complete, attempting to copy data back to Host\n";
    checkCuda( hipMemcpy(h_mean,d_mean,2048*sizeof(double),hipMemcpyDeviceToHost) );

    /* This loop is meant for testing the validity of the memcpy output
    for(int i=0;i<2048;++i){
        std::cout<<"element "<<i<<" : "<<h_mean[i]<<std::endl;
    }
    */

    // Standard Error of the bootstrap means
    int n = 2048;
    double SumOfMeans=0;
    for(int i=0;i<n;i++){
	SumOfMeans+=h_mean[i];
    }
    double MeanOfMeans = SumOfMeans / (double)n;
    std::cout<<"MeanofMeans: "<<MeanOfMeans<<"\n";

    double SqrDiff=0;
    for(int i=0;i<n;i++){
	SqrDiff += (h_mean[i]-MeanOfMeans) * (h_mean[i]-MeanOfMeans);
    }
    double SqrdVariance = SqrDiff / (n-1);
    double BootError = ((n-1)/(n*n)) * SqrdVariance;
    std::cout<<"SqrDiff, SqrdVariance, & BootError: "<<SqrDiff<<", "<<SqrdVariance<<", "<<BootError<<"\n";

    //std::sort(h_mean,h_mean+2048);
    std::cout<<"\nStandard Error is: "<<BootError<<"\n\n\n";

    double C_Arr[2048] = {};
    for(int i=0;i<n;i++){
	C_Arr[i]=h_mean[i]-BaseMean;
    }
    std::sort(std::begin(C_Arr),std::end(C_Arr));
    double L = 2048.0 * 0.1;
    int Lower = (int)L;
    double H = 2048 * 0.9;
    int Higher = (int)H;

    int LowerBound = C_Arr[Lower];
    int UpperBound = C_Arr[Higher];

    double Left = BaseMean - (double)LowerBound;
    double Right = BaseMean - (double)UpperBound;

    std::cout<<"\n\n\n------------------------------------------------\n";
    std::cout<<"The Confidence Interval is: 80%\n";
    std::cout<<"The Standard Error is: "<<BootError<<"\n";
    std::cout<<"This is on the 10th & 90th percentiles: ["<<Left<<", "<<Right<<"]\n";

    checkCuda( hipFree(d_Base) );
    checkCuda( hipFree(d_mean) );
    //checkCuda( cudaFree(devStates) );
    checkCuda( hipHostFree(BaseSample) );
    checkCuda( hipHostFree(h_mean) );
    printf("\n\n\nDONE\n");


    return 0;
}









//////////////////// Calculate Statistics. Soon to be offloaded
// int finalMean[400]={0};
// // std::vector<int> *meanVector;
// int bnum = 0;
// int sum1;
// int sum2 = 0;
// // int temp1;
// for (int a=0;a<400;++a){
//     sum1=0;
//     for(int b=0;b<100;++b){
//         sum1+=h_mean[b+(100*bnum)];
//     }
//     finalMean[a]=sum1/100;
//     // temp1 = sum1/100;
//     // meanVector[a].push_back( temp1 );
//     sum2 += std::pow( (finalMean[a]-meanOriginal), 2 );
//     bnum++;
//     // std::cout<<"Final Mean "<<a<<" : "<<finalMean[a]<<std::endl;
// }
// printf("\n\n\n");
// std::sort(finalMean,finalMean+SAMPLE_SIZE);
// std::cout<<"sum2 is "<<sum2<<std::endl;
// int div = 400;
// std::cout<<"div is "<<div<<std::endl;
// float stdDeviation = sqrt( (sum2/div) );
// std::cout<<"Standard Deviation is "<<stdDeviation<<std::endl;
// float stdErrorFactor = ( 100.0 / (100.0-1.0) );
// std::cout<<"The Error Factor is "<<stdErrorFactor<<std::endl;
// float stdError = sqrt( stdErrorFactor ) * stdDeviation;
// std::cout<<"Standard Error is "<<stdError<<std::endl;
// int tempA; int tempB;
// float lowerCI = 400 * ( 0.05/2 );
// tempA = finalMean[(int)lowerCI];
// std::cout<<"Lower (5%) Confidence Interval is "<<tempA<<std::endl;
// float higherCI = 400 * ( 1 - (0.05/2) );
// tempB = finalMean[(int)higherCI];
// std::cout<<"Higher (95%) Confidence Interval is "<<tempB<<std::endl;
